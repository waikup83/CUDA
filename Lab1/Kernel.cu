// David Desbiens
// Application qui traite l'arri�re plan d'une image

//Cuda
#include "hip/hip_runtime.h"


#include <iostream>


//Structure qui contient le nombre d'�l�ment dans le tableau et des diff�rentes �cart � appliquer
struct InfoImage {
	int Compteur;
	int NEcartB;
	int NEcartV;
	int NEcartR;
};


extern "C" hipError_t Analyser(unsigned char *Image, int x, int y, unsigned char *Echantillons, InfoImage InfoImg, char Type);



//Kernel pour enlever le fond
__global__ void ChangerFond(unsigned char *Addr, unsigned char *Tab, InfoImage InfoImg)
{
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
	int min = 0, max = 0;

	for (int x = 0; x < InfoImg.Compteur; x += 6) {
		//Calcule du min Bleu
		min = Tab[x] - Tab[x + 1] * InfoImg.NEcartB;
		if (min < 0)
			min = 0;
		//Calcule du max Bleu
		max = Tab[x] + Tab[x + 1] * InfoImg.NEcartB;
		if (min > 255)
			min = 255;


		if (Addr[i] >= min && Addr[i] <= max)
		{
			//Calcule du min Vert
			min = Tab[x + 2] - Tab[x + 3] * InfoImg.NEcartV;
			if (min < 0)
				min = 0;
			//Calcule du max Vert
			max = Tab[x + 2] + Tab[x + 3] * InfoImg.NEcartV;
			if (min > 255)
				min = 255;


			if (Addr[i + 1] >= min && Addr[i + 1] <= max)
			{
				//Calcule du min Rouge
				min = Tab[x + 4] - Tab[x + 5] * InfoImg.NEcartR;
				if (min < 0)
					min = 0;
				//Calcule du max Rouge
				max = Tab[x + 4] + Tab[x + 5] * InfoImg.NEcartR;
				if (min > 255)
					min = 255;

				//Mettre en blanc si rentre dans toutes les conditions
				if (Addr[i + 2] >= min && Addr[i + 2] <= max)
				{
					Addr[i] = 255;
					Addr[i + 1] = 255;
					Addr[i + 2] = 255;
				}

			}
		}
	}
}


//Kernel pour appliquer le sobel
__global__ void AppliquerSobel(unsigned char *Addr, unsigned char *Addr2, int Largeur, int gImage) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float SommeX = 0;
	float SommeY = 0;

	Largeur *= 3;


	//Gradient en x
	//Pixels du haut
	if (&Addr[i - Largeur - 3] >= &Addr[0])
		SommeX += (Addr[i - Largeur - 3] * -1);
	if (&Addr[i - Largeur + 3] >= &Addr[0])
		SommeX += (Addr[i - Largeur + 3]);
	//Pixels du centre
	if (i / Largeur == (i - 3) / Largeur)
		SommeX += (Addr[i - 3] * -2);
	if (i / Largeur == (i + 3) / Largeur)
		SommeX += (Addr[i + 3] * 2);
	//Pixels du bas
	if (&Addr[i + Largeur - 3] <= &Addr[gImage])
		SommeX += (Addr[i + Largeur - 3] * -1);
	if (&Addr[i + Largeur + 3] <= &Addr[gImage])
		SommeX += (Addr[i + Largeur + 3]);

	SommeX = SommeX * SommeX;


	//Gradient en y
	//Pixels du haut
	if (&Addr[i - Largeur - 3] >= &Addr[0])
		SommeY += (Addr[i - Largeur - 3] * -1);
	if (&Addr[i - Largeur] >= &Addr[0])
		SommeY += (Addr[i - Largeur] * -2);
	if (&Addr[i - Largeur + 3] >= &Addr[0])
		SommeY += (Addr[i - Largeur + 3] * -1);
	//Pixels du bas
	if (&Addr[i + Largeur - 3] <= &Addr[gImage])
		SommeY += (Addr[i + Largeur - 3]);
	if (&Addr[i + Largeur] <= &Addr[gImage])
		SommeY += (Addr[i + Largeur] * 2);
	if (&Addr[i + Largeur + 3] <= &Addr[gImage])
		SommeY += (Addr[i + Largeur + 3]);

	SommeY = SommeY * SommeY;
	

	Addr2[i] = sqrt(SommeX + SommeY);
}



/* Type
		0 : enl�ve la couleur du fond
		1 : applique sobel
*/
hipError_t Analyser(unsigned char *Image, int x, int y, unsigned char *Tableau, InfoImage InfoImg, char Type)
{
	hipError_t Erreur;
	unsigned char *AddrGPU = 0;
	unsigned char *AddrGPU2 = 0;
	unsigned char *AddrTab = 0;


	//S�lectionner le GPU
	Erreur = hipSetDevice(0);
	if (Erreur != hipSuccess)
	{
		std::cout << "S�lection de la carte impossible\r\n";
		return Erreur;
	}

	//R�servation de la m�moire GPU pour l'image
	Erreur = hipMalloc((void**)&AddrGPU, x * y * 3);
	if (Erreur != hipSuccess)
	{
		std::cout << "Allocation de la m�moire impossible\r\n";
		return Erreur;
	}

	//Copie vers la m�moire GPU de l'image
	Erreur = hipMemcpy(AddrGPU, Image, x * y * 3, hipMemcpyHostToDevice);
	if (Erreur != hipSuccess)
	{
		std::cout << "Copie vers le GPU impossible\r\n";
		return Erreur;
	}




	//**********************************************************************************
	//Enlever la couleur de fond
	if (Type == 0) {
		//Allocation m�moire pour le tableau des �chantillons
		Erreur = hipMalloc((void**)&AddrTab, InfoImg.Compteur);
		if (Erreur != hipSuccess)
		{
			std::cout << "Allocation de la m�moire impossible\r\n";
			return Erreur;
		}

		//Copie vers la m�moire GPU de l'image
		Erreur = hipMemcpy(AddrTab, Tableau, InfoImg.Compteur, hipMemcpyHostToDevice);
		if (Erreur != hipSuccess)
		{
			std::cout << "Copie vers le GPU impossible\r\n";
			return Erreur;
		}
		//Calcul du nombre de thread  ... 960 par bloc
		ChangerFond<<<x * y / 960, 960>>>(AddrGPU, AddrTab, InfoImg);
	}

	//Appliquer un sobel
	else if (Type == 1)
	{
		//R�servation de la m�moire GPU pour l'image r�sultante en x
		Erreur = hipMalloc((void**)&AddrGPU2, x * y * 3);
		if (Erreur != hipSuccess)
		{
			std::cout << "Allocation de la m�moire impossible\r\n";
			return Erreur;
		}

		AppliquerSobel<<<x * y * 3 / 960, 960>>>(AddrGPU, AddrGPU2, x, x * y * 3);
	}
	//**********************************************************************************
    
	
	
	
	//V�rifier si erreur pendant l'ex�cution du code
	Erreur = hipGetLastError();
    if (Erreur != hipSuccess) {
		std::cout << "Erreur d'ex�cution GPU : " << hipGetErrorString(Erreur) << "\r\n";
		return Erreur;
    }
    
	//Synchroniser avec le GPU
	Erreur = hipDeviceSynchronize();
    if (Erreur != hipSuccess) {
		std::cout << "Erreur de synchronisation : " << hipGetErrorString(Erreur) << "\r\n";
		return Erreur;
    }




	//******************************************************************
    //Copie des donn�es GDDR vers DDR
	if (Type == 0) {
		Erreur = hipMemcpy(Image, AddrGPU, x * y * 3, hipMemcpyDeviceToHost);
		if (Erreur != hipSuccess) {
			std::cout << "Copie vers le CPU impossible\r\n";
			return Erreur;
		}
	}
	else if (Type == 1) {
		Erreur = hipMemcpy(Image, AddrGPU2, x * y * 3, hipMemcpyDeviceToHost);
		if (Erreur != hipSuccess) {
			std::cout << "Copie vers le CPU impossible\r\n";
			return Erreur;
		}
	}
	//*******************************************************************




	//Vidage du GPU
	Erreur = hipDeviceReset();
	if (Erreur != hipSuccess) {
		std::cout << "M�nage de la m�moire impossible\r\n";
		return Erreur;
	}


	return Erreur;
}
